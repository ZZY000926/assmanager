#include <hip/hip_runtime.h>

__global__ void calx_kernel(double* xens, double* zens_wrap, double* a, int ensemble_size, int model_size, int ss2, int smooth_steps) 
{
    int iens = blockIdx.x * blockDim.x + threadIdx.x;

    if (iens >= ensemble_size)
    {
        return;
    }

    double *x = xens + iens * model_size;
    double *zwrap = zens_wrap + iens * model_size;

    for (int i = ss2; i < ss2 + model_size; i++)
    {
        x[i - ss2] = a[0] * zwrap[i + 1 - (- smooth_steps)] / 2.00;
        for (int j = -smooth_steps + 1; j < smooth_steps; j++)
        {
            x[i - ss2] = x[i - ss2] + a[j + smooth_steps] * zwrap[i + 1 - j];
        }
        x[i - ss2] = x[i - ss2] + a[2 * smooth_steps] * zwrap[i + 1 - smooth_steps] / 2.00;
    }
}


__global__ void calw(double* wxens, double* xens_wrap, int ensemble_size, int model_size, int K, int K4, int H) 
{
    int iens = blockIdx.x * blockDim.x + threadIdx.x;

    if (iens >= ensemble_size)
    {
        return;
    }

    double *wx = wxens + iens * model_size;
    double *xwrap = xens_wrap + iens * model_size;

    for (int i = K4; i < K4 + model_size; i++)
    {
        wx[i] = xwrap[i - (-H)] / 2.00;
        for (int j = -H + 1; j < H; j++)
            wx[i] = wx[i] + xwrap[i - j];

        wx[i] = wx[i] + xwrap[i - H] / 2.00;
        wx[i] = wx[i] / K;
    }
}

     
__global__ void caldz(double* wxens, double* xens_wrap, double* dzens, double* yens_wrap,
          int ensemble_size, int model_size, double space_time_scale, double sts2, double coupling, 
          double forcing, int K, int K2, int K4, int H, int model_number) 
{
    int iens = blockIdx.x * blockDim.x + threadIdx.x;

    if (iens >= ensemble_size)
    {
        return;
    }

    double *wx = wxens + iens * model_size;
    double *xwrap = xens_wrap + iens * model_size;
    double *dz = dzens + iens * model_size;
    double *ywrap = yens_wrap + iens * model_size;

    for (int i = K4; i < K4 + model_size; i++)
    {
        double xx = wx[i - K + (-H)] * xwrap[i + K + (-H)] / 2.00;
        for (int j = -H + 1; j < H; j++)
        {
            xx = xx + wx[i - K + j] * xwrap[i + K + j];
        }
        xx = xx + wx[i - K + H] * xwrap[i + K + H] / 2.00;
        xx = - wx[i - K2] * wx[i - K] + xx / K;

        if (model_number == 3)
        {
            dz[i - K4] = xx + sts2 * (- ywrap[i - 2] * ywrap[i - 1] + ywrap[i - 1] * ywrap[i + 1])
                            + coupling * (- ywrap[i - 2] * xwrap[i - 1] + ywrap[i - 1] * xwrap[i + 1]) - xwrap[i]
                            - space_time_scale * ywrap[i] + forcing;
        }
        else // model II
        {
            dz[i - K4] = xx - xwrap[i] + forcing;
        }
    }
}


#ifdef __cplusplus
extern "C"
#endif

void run_calx_kernel(double* xens, double* zens_wrap, double* a, int ensemble_size, int model_size, int ss2, int smooth_steps)
{
    int block_size = 256;
    int grid_size = (ensemble_size + block_size - 1) / block_size;

    calx_kernel<<<grid_size, block_size>>>(xens, zens_wrap, a, ensemble_size, model_size, ss2, smooth_steps);
}


void run_calw_kernel(double* wxens, double* xens_wrap, int ensemble_size, int model_size, int K, int K4, int H)
{
    int block_size = 256;
    int grid_size = (ensemble_size + block_size - 1) / block_size;

    calw<<<grid_size, block_size>>>(wxens, xens_wrap, ensemble_size, model_size, K, K4, H);
}


void run_caldz_kernel(double* wxens, double* xens_wrap, double* dzens, double* yens_wrap,
          int ensemble_size, int model_size, double space_time_scale, double sts2, double coupling, 
          double forcing, int K, int K2, int K4, int H, int model_number)
{
    int block_size = 256;
    int grid_size = (ensemble_size + block_size - 1) / block_size;

    caldz<<<grid_size, block_size>>>(wxens, xens_wrap, dzens, yens_wrap,
          ensemble_size, model_size, space_time_scale, sts2, coupling, 
          forcing, K, K2, K4, H, model_number);
}